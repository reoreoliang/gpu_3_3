#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

// ���´�����Ϊ����ʾ�����ڴ���߳�ͬ����ʹ��
// ��Ч���Ǽ����ֵ
__global__ void gpu_shared_memory(float* d_a) {
	int i, index = threadIdx.x;  // ������0��ʼ
	float average, sum = 0.0f;
	// ���干���ڴ�
	__shared__ float sh_arr[10];
	sh_arr[index] = d_a[index];  // �����ݴ�ȫ���ڴ�д�뵽�����ڴ���

	// �������ָ���������ȷ���ڼ���ִ�г���֮ǰ����ɶ��ڴ������д�����
	__syncthreads();
	for (i = 0; i <= index; i++)  // ����ѭ��
	{
		sum += sh_arr[i];
	}
	average = sum / (index + 1.0f);
	d_a[index] = average;

	sh_arr[index] = average;  // ����ֻ��Ϊ����ʾһ�¹����ڴ���������ڣ�ע������ע
}

int main(void)
{
	float h_a[10], * d_a;
	// ��ֵ��ֵ
	for (int i = 0; i < 10; i++)
	{
		h_a[i] = i;
	}

	hipMalloc((void **)&d_a, 10 * sizeof(float));
	hipMemcpy((void *)d_a, (void *)h_a, 10 * sizeof(float), hipMemcpyHostToDevice);
	gpu_shared_memory << <1, 10 >> > (d_a);
	hipMemcpy((void*)h_a, (void*)d_a, 10 * sizeof(float), hipMemcpyDeviceToHost);

	printf("Use of shared Memory on GPU\n");
	for (int i = 0; i < 10; i++)
	{
		printf("The running average after %d element is %f \n", i, h_a[i]);
	}
	return 0;
}